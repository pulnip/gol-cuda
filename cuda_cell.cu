#include "hip/hip_runtime.h"
#include <stdio.h>
#include <glad/glad.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "cuda_entry.h"
#include "cuda_util.cuh"

constexpr int WIDTH = 800;
constexpr int HEIGHT = 600;
constexpr int SIZE = WIDTH * HEIGHT;

static unsigned char* dev_prev = nullptr;
static unsigned char* dev_curr = nullptr;

static hipGraphicsResource* cudaTexResource = nullptr;

void initCellGrid(int width, int height) {
    if (dev_prev || dev_curr) return;

    checkCuda(hipMalloc(&dev_prev, SIZE));
    checkCuda(hipMalloc(&dev_curr, SIZE));

    unsigned char* h_init = new unsigned char[SIZE];
    for (int i = 0; i < SIZE; ++i)
        h_init[i] = (rand() & 1) ? 255 : 0; // use 0 or 255 so GL_R8 shows visible white/black

    checkCuda(hipMemcpy(dev_prev, h_init, SIZE, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(dev_curr, h_init, SIZE, hipMemcpyHostToDevice));
    delete[] h_init;
}

void destroyCuda() {
    checkCuda(hipFree(dev_prev));
    checkCuda(hipFree(dev_curr));
    dev_prev = nullptr;
    dev_curr = nullptr;

    checkCuda(hipDeviceReset());
}

void registerCudaTexture(GLuint texture){
    glBindTexture(GL_TEXTURE_2D, texture);
    glFinish();
    checkCuda(hipGraphicsGLRegisterImage(&cudaTexResource,
        texture, GL_TEXTURE_2D,
        hipGraphicsRegisterFlagsWriteDiscard));
}

void updateCellTexture() {
    hipArray_t array;
    checkCuda(hipGraphicsMapResources(1, &cudaTexResource));
    checkCuda(hipGraphicsSubResourceGetMappedArray(&array,
        cudaTexResource, 0, 0));

    const size_t widthBytes = WIDTH * sizeof(unsigned char);
    checkCuda(hipMemcpy2DToArray(array, 0, 0, dev_prev,
        widthBytes, widthBytes, HEIGHT, hipMemcpyDeviceToDevice));

    checkCuda(hipGraphicsUnmapResources(1, &cudaTexResource));
}

__global__ void updateKernel(unsigned char* current, unsigned char* next, int width, int height) {
int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int count = 0;
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            if (dx == 0 && dy == 0) continue;
            int nx = x + dx;
            int ny = y + dy;
            if (nx >= 0 && ny >= 0 && nx < width && ny < height) {
                int idx = ny * width + nx;
                count += (current[idx] > 0); // 255이면 살아있음
            }
        }
    }

    int idx = y * width + x;
    if (current[idx]) {
        // 살아있는 셀
        next[idx] = (count == 2 || count == 3) ? 255 : 0;
    } else {
        // 죽은 셀
        next[idx] = (count == 3) ? 255 : 0;
    }
}

void updateCell() {
    dim3 threads(16, 16);
    dim3 blocks((WIDTH + 15)/16, (HEIGHT + 15)/16);

    updateKernel<<<blocks, threads>>>(dev_prev, dev_curr, WIDTH, HEIGHT);
    std::swap(dev_prev, dev_curr);
}

unsigned char* getCurrentCellBuffer() {
    return dev_prev;
}